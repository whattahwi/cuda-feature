#include <stdio.h>
#include <hip/hip_runtime.h>

// A utility function to check CUDA errors
void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error: %s: %s\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

// A utility function to get the device properties
void getDeviceProperties(int device, hipDeviceProp_t* prop) {
    hipError_t err = hipGetDeviceProperties(prop, device);
    checkCudaError(err, "hipGetDeviceProperties");
}

// A utility function to get the SM count of the device
int getSMCount(int device) {
    hipDeviceProp_t prop;
    getDeviceProperties(device, &prop);
    return prop.multiProcessorCount;
}

// A utility function to get the core count per SM of the device
int getCoreCountPerSM(int device) {
    hipDeviceProp_t prop;
    getDeviceProperties(device, &prop);
    int cores = 0;
    switch (prop.major) {
        case 3: // Kepler
            cores = 192;
            break;
        case 5: // Maxwell
            cores = 128;
            break;
        case 6: // Pascal
            if (prop.minor == 0) cores = 64; 
            else if (prop.minor == 1) cores = 128;
            else if (prop.minor == 2) cores = 128;
            break;
        case 7: // Volta
            if (prop.minor == 0) cores = 64;
            else if (prop.minor == 5) cores = 64;
            break;
        case 8: // Ampere
            if (prop.minor == 0) cores = 64;
            else if (prop.minor == 6) cores = 128;
            break;
        default:
            fprintf(stderr, "Unknown device architecture: %d.%d\n", prop.major, prop.minor);
            exit(EXIT_FAILURE);
    }
    return cores;
}

// A utility function to get the core frequency in KHz of the device
int getCoreFrequency(int device) {
    hipDeviceProp_t prop;
    getDeviceProperties(device, &prop);
    return prop.clockRate;
}

// A utility function to get the peak FLOPS of the device
// This is based on the core count, core frequency, tensor core count, tensor core frequency, and tensor core throughput
double getPeakFlops(int device, hipDataType type) {
    int sm_count = getSMCount(device); // get the number of SMs
    int core_count_per_sm = getCoreCountPerSM(device); // get the number of cores per SM
    int core_frequency = getCoreFrequency(device); // get the core frequency in MHz
    double peak_flops = 0.0; // initialize the peak FLOPS
    // calculate the peak FLOPS based on the data type
    switch (type) {
        case HIP_R_16F: // FP16
            peak_flops = (double)sm_count * core_count_per_sm * core_frequency * 1e3 * 4;
            break;
        case HIP_R_32F: // FP32
            peak_flops = (double)sm_count * core_count_per_sm * core_frequency * 1e3 * 2;
            break;
        default:
            fprintf(stderr, "Unsupported data type for peak FLOPS: %d\n", type);
            exit(EXIT_FAILURE);
    }
    return peak_flops;
}

// A main function to test the peak FLOPS of the device
int main() {
    int device = 0; // use the first device by default
    hipDataType type = HIP_R_32F; // use FP16 data type by default
    
    // get the device name
    hipDeviceProp_t prop;
    getDeviceProperties(device, &prop);
    printf("Device name: %s\n", prop.name);
    // get the peak FLOPS
    double peak_flops = getPeakFlops(device, type);
    printf("Peak FLOPS: %.2f GFLOPS\n", peak_flops / 1e9);
    return 0;
}
